#include "hip/hip_runtime.h"
#include "blur.h"

#define TILE_WIDTH 16

__global__ void blurKernel(float *out, float *in, int width, int height, int blurSize)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int contextRadius = (blurSize - 1) / 2;
    float accumVal = 0.0f;
    int count = 0;

    if (x >= contextRadius && x < width - contextRadius && y >= contextRadius && y < height - contextRadius)
    {
        for (int dy = -contextRadius; dy <= contextRadius; dy++)
        {
            for (int dx = -contextRadius; dx <= contextRadius; dx++)
            {
                int ix = x + dx;
                int iy = y + dy;
                int index = iy * width + ix;

                accumVal += in[index];
                count++;
            }
        }

        out[y * width + x] = accumVal / count;
    }
    else
    {
        out[y * width + x] = 0.0f;
    }

}



Image imageBlurOnDevice(const Image &inputImage, int blurSize)
{
    Image outputImage(inputImage.getWidth(), inputImage.getHeight(), true);

    // allocate input and output images in the device
    float *d_inputImage;
    float *d_outputImage;
    hipMalloc((void **)&d_inputImage, inputImage.getRows() * inputImage.getCols() * sizeof(float));
    hipMalloc((void **)&d_outputImage, outputImage.getRows() * outputImage.getCols() * sizeof(float));

    hipMemcpy(d_inputImage, inputImage.getDataConstPtr(), inputImage.getRows() * inputImage.getCols() * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimGrid(ceil((float)outputImage.getCols() / TILE_WIDTH),
                ceil((float)outputImage.getRows() / TILE_WIDTH));
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    blurKernel<<<dimGrid, dimBlock>>>(d_outputImage, d_inputImage,
        outputImage.getCols(), outputImage.getRows(), blurSize);

    hipMemcpy(outputImage.getDataPtr(), d_outputImage, outputImage.getRows() * outputImage.getCols() * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_inputImage);
    hipFree(d_outputImage);

    return outputImage;
}

Image imageBlurOnHost(const Image &inputImage, int blurSize)
{
    Image outputImage(inputImage.getWidth(), inputImage.getHeight(), inputImage.isGray());

    int contextRadius = (blurSize - 1) / 2;

    for (unsigned int y = 0; y < inputImage.getHeight(); ++y)
    {
        for (unsigned int x = 0; x < inputImage.getWidth(); ++x)
        {
            float outVal = 0.0f;
            // Inside full context space
            if (x >= contextRadius && x < inputImage.getWidth() - contextRadius && y >= contextRadius && y < inputImage.getHeight() - contextRadius)
            {
                float accumVal = 0.0f;
                for (int c = -contextRadius; c <= contextRadius; c++)
                {
                    for (int r = -contextRadius; r <= contextRadius; ++r)
                    {
                        int accumIdx = (y + c) * inputImage.getWidth() + (x + r);
                        accumVal += inputImage.getDataConstPtr()[accumIdx];
                    }
                }
                outVal = accumVal / (blurSize * blurSize);
            }

            int outIdx = y * inputImage.getWidth() + x;
            outputImage.getDataPtr()[outIdx] = outVal;
        }
    }

    return outputImage;
}
